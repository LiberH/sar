
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>

#define FALSE 0
#define TRUE  !FALSE
#define DEFAULT_N 1000000000

struct data {
  char *list;
  int *primes;
  int count;
  long double time;
};

int GPU_process (struct data *gpu, int n);
int CPU_process (struct data *cpu, int n);

int
main (int argc, char **argv)
{
  int n, i, failed;
  struct data gpu, cpu;

  n = DEFAULT_N;
  if (argc > 1)
    n = atoi (argv[1]);

  /* GPU: */
  gpu.count = GPU_process (&gpu, n);
  printf ("GPU: %d primes to %d\n", gpu.count, n);
  printf ("CPU: %Lf seconds\n", gpu.time);

  /* CPU: */
  cpu.count = CPU_process (&cpu, n);
  printf ("CPU: %d primes to %d\n", cpu.count, n);
  printf ("CPU: %Lf seconds\n", cpu.time);

  /* Check: */
  failed = FALSE;
  for (i = 0; i < n; ++i)
    if (gpu.primes[i] != cpu.primes[i])
      {
	failed = TRUE;
	printf ("      %d <> %d (%d)\n", gpu.primes[i], cpu.primes[i], (i +1));
      }

  if (failed)
    printf (" ! : Results differ.\n");

  return EXIT_SUCCESS;
}

void
CPU_init (int **primes, char **list, int n)
{
  *list = (char *) malloc (sizeof (char) * (n +1));
  if (*list == NULL)
    {
      perror ("malloc");
      exit (EXIT_FAILURE);
    }

  *primes = (int *) malloc (sizeof (int) * (n +1));
  if (*primes == NULL)
    {
      perror ("malloc");
      exit (EXIT_FAILURE);
    }
}

void
CPU_sieve (char *list, int n)
{
  int s, k, i;

  s = (int) sqrt (n);
  for (k = 2; k < (s +1); ++k)

    /* Check if not marked: */
    if (!*(list + k))

      /* Mark each multiple: */
      for (i = k * k; i < (n +1); i += k)
	*(list + i) = TRUE;
}

int
CPU_store (int *primes, char *list, int n)
{
  int p, k;

  p = 0;
  for (k = 2; k < (n +1); ++k)

    /* Check if not marked: */
    if (!*(list + k))
      *(primes + p++) = k;

  return p++;
}

int
CPU_process (struct data *cpu, int n)
{
  int p, err;
  struct timeval a, b;

  CPU_init (&cpu -> primes, &cpu -> list, n);

  err = gettimeofday (&a, NULL);
  if (err)
    {
      perror ("gettimeofday");
      exit (EXIT_FAILURE);
    }

  CPU_sieve (cpu -> list, n);
  p = CPU_store (cpu -> primes, cpu -> list, n);

  err = gettimeofday (&b, NULL);
  if (err)
    {
      perror ("gettimeofday");
      exit (EXIT_FAILURE);
    }

  cpu -> time = (b.tv_sec + (b.tv_usec * 1.0e-6L)) -
    (a.tv_sec + (a.tv_usec * 1.0e-6L));
  return p;
}

void
GPU_init (char **cuda_list, int n)
{
  hipError_t cuda_err;

  cuda_err = hipMalloc (cuda_list, sizeof (char) * (n +1));
  if (cuda_err != hipSuccess)
    {
      fprintf (stderr, "cudaMalloc: %s\n", hipGetErrorString (cuda_err));
      exit (EXIT_FAILURE);
    }

  cuda_err = hipMemset (*cuda_list, 0, sizeof (char) * (n +1));
  if (cuda_err != hipSuccess)
    {
      fprintf (stderr, "cudaMemset: %s\n", hipGetErrorString (cuda_err));
      exit (EXIT_FAILURE);
    }

}

__global__ void
kkk (char *list, int n)
{
  int k;

  k = blockDim.x * blockIdx.x + threadIdx.x;
  printf ("%d. %d\n", k, *(list + k));
  *(list + k) = k;
}

__global__ void
kernel (char *list, int n, int s)
{
  int k, i;

  k = blockDim.x * blockIdx.x + threadIdx.x;
  if (k > 1 && k < (s +1))

    /* Check if not marked: */
    if (!*(list + k))

      /* Mark each multiple: */
      for (i = k * k; i < (n +1); i += k)
	*(list + i) = TRUE;
}

void
GPU_sieve (char *list, char *cuda_list, int n)
{
  int s;
  hipError_t cuda_err;
  
  s = (int) sqrt (n);
  kernel <<<(s/32 +1), (32)>>> (cuda_list, n, s);
  cuda_err = hipDeviceSynchronize ();
  if (cuda_err != hipSuccess)
    {
      fprintf (stderr, "cudaDeviceSynchronize: %s\n", hipGetErrorString (cuda_err));
      exit (EXIT_FAILURE);
    }

  cuda_err = hipMemcpy (list, cuda_list, sizeof (char) * (n +1), hipMemcpyDeviceToHost);
  if (cuda_err != hipSuccess)
    {
      fprintf (stderr, "cudaMemcpy: %s\n", hipGetErrorString (cuda_err));
      exit (EXIT_FAILURE);
    }
}

int
GPU_process (struct data *gpu, int n)
{
  int p, err;
  char *cuda_list = NULL;
  struct timeval a, b;

  CPU_init (&gpu -> primes, &gpu -> list, n);
  GPU_init (&cuda_list, n);

  err = gettimeofday (&a, NULL);
  if (err)
    {
      perror ("gettimeofday");
      exit (EXIT_FAILURE);
    }

  GPU_sieve (gpu -> list, cuda_list, n);
  p = CPU_store (gpu -> primes, gpu -> list, n);

  err = gettimeofday (&b, NULL);
  if (err)
    {
      perror ("gettimeofday");
      exit (EXIT_FAILURE);
    }

  gpu -> time = (b.tv_sec + (b.tv_usec * 1.0e-6L)) -
    (a.tv_sec + (a.tv_usec * 1.0e-6L));
  return p;
}
